#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include "octree.h"

#define COORD_MAX 40.0f
#define COORD_MIN -40.0f

FLOAT3 * unc_pos;
FLOAT3 * dev_buffer1;
FLOAT3 * dev_buffer2;
Octree_node * dev_nodes;
const int THREADS_PER_BLOCK = 128;
const int shared_mem = 16*sizeof(int);

__host__ static __inline__ void rnd()
{
  for (int i = 0; i < nbody; i++) {
    unc_pos[i].x = COORD_MIN + (rand() / ( RAND_MAX / (COORD_MAX-COORD_MIN) ) ) ;
    unc_pos[i].y = COORD_MIN + (rand() / ( RAND_MAX / (COORD_MAX-COORD_MIN) ) ) ;
    unc_pos[i].z = COORD_MIN + (rand() / ( RAND_MAX / (COORD_MAX-COORD_MIN) ) ) ;
  }
}

__device__ bool check_points(Octree_node &node, FLOAT3 *points1, FLOAT3 *points2, int num_points, Parameters params){
  if(params.depth >= params.max_depth || num_points <= params.min_points){
    if(params.point_selector == 1){
      int it = node.points_begin(); int end = node.points_end();
      for(it += threadIdx.x; it < end; it += blockDim.x){
        points1[it] = points2[it];
        // points[0].set_point(it, points[1].get_point(it));
      }
    }
    return true;
  }
  return false;
}

__device__ void count_points(const FLOAT3 *in_points, int *smem, int range_begin, int range_end, FLOAT3 center){
  if(threadIdx.x < 8) smem[threadIdx.x] = 0;
  __syncthreads();

  for(int iter=range_begin+threadIdx.x; iter<range_end; iter+=blockDim.x){
    FLOAT3 p = in_points[iter];

    int x = p.x < center.x ? 0 : 1;
    int y = p.y < center.y ? 0 : 1;
    int z = p.z < center.z ? 0 : 1;

    int i = x*4 + y*2 + z;

    atomicAdd(&smem[i], 1);
  }
  __syncthreads();
}

__device__ void scan_offsets(int node_points_begin, int* smem){
  int *smem2 = &smem[8];
  if(threadIdx.x == 0){
    for(int i = 0; i < 8; i++){
      smem2[i] = i == 0 ? 0 : smem2[i-1] + smem[i-1];
    }
    for (int i = 0; i < 8; i++){
      smem2[i] += node_points_begin;
    }
  }
  __syncthreads();
}

__device__ void reorder_points(FLOAT3 *out_points, const FLOAT3 *in_points, int *smem, int range_begin, int range_end, FLOAT3 center){
  int *smem2 = &smem[8];

  for(int iter = range_begin+threadIdx.x; iter<range_end; iter+=blockDim.x){
    FLOAT3 p = in_points[iter];

    int x = p.x < center.x ? 0 : 1;
    int y = p.y < center.y ? 0 : 1;
    int z = p.z < center.z ? 0 : 1;

    int i = x*4 + y*2 + z;

    int dest = atomicAdd(&smem2[i], 1);
    out_points[dest] = p;
  }

  __syncthreads();
}

__device__ void prepare_children(Octree_node *children, Octree_node &node, int *smem){

  int child_offset = 8*node.id();

  for(int i = 0; i < 8; i++){
    children[child_offset+i].set_id(8*node.id()+(i*8));
  }

  const FLOAT3 center = node.center();
  float half = node.width() / 2.0f;
  float quarter = half / 2.0f;

  for(int i = 0; i < 8; i++){
    float xf, yf, zf;
    xf = i / 4 == 0 ? -1.0f : 1.0f;
    yf = (i-4) / 4 == 0 ? -1.0f : 1.0f;
    zf = i % 2 == 0 ? -1.0f : 1.0f;

    children[child_offset+i].set_center(center.x + quarter * xf,
                                        center.y + quarter * yf,
                                        center.z + quarter * zf);

    children[child_offset+i].set_width(half);
    children[child_offset+i].set_range(smem[8+i], smem[i]+smem[8+i]);
  }
}

__global__ void build_octree_kernel(Octree_node *nodes, FLOAT3 *points1, FLOAT3 *points2, Parameters params){
  __shared__ int smem[16];

  Octree_node &node = nodes[blockIdx.x];
  node.set_id(node.id() + blockIdx.x);
  int num_points = node.num_points();

  bool exit = check_points(node, points1, points2, num_points, params);
  if(exit) return;

  float3 center = node.center();

  int range_begin = node.points_begin();
  int range_end = node.points_end();
  // const Points &in_points = points[params.point_selector];
  const FLOAT3* in_points = params.point_selector == 0 ? points1 : points2;
  // Points &out_points = points[(params.point_selector + 1) % 2];
  FLOAT3 *out_points = params.point_selector == 0 ? points2 : points1;

  count_points(in_points, smem, range_begin, range_end, center);

  scan_offsets(node.points_begin(), smem);

  reorder_points(out_points, in_points, smem, range_begin, range_end, center);

  if(threadIdx.x == blockDim.x-1){
    Octree_node *children = &nodes[params.nodes_in_level];
    prepare_children(children, node, smem);
    build_octree_kernel<<<8, blockDim.x, 16*sizeof(int)>>>(children, points1, points2, Parameters(params, true));
  }
}

int main(){

  unc_pos = new FLOAT3[nbody];
  rnd();

  hipMalloc((void**) &dev_buffer1, nbody*sizeof(FLOAT3));
  hipMalloc((void**) &dev_buffer2, nbody*sizeof(FLOAT3));
  hipMemcpy(dev_buffer1, unc_pos, nbody*sizeof(FLOAT3), hipMemcpyHostToDevice);

  Octree_node root;
  root.set_range(0, nbody);
  root.set_width(2*COORD_MAX);
  hipMalloc((void **)&dev_nodes, nbody*sizeof(Octree_node));
  hipMemcpy(dev_nodes, &root, sizeof(Octree_node), hipMemcpyHostToDevice);

  Parameters params(nbody);
  build_octree_kernel<<<1, THREADS_PER_BLOCK, shared_mem>>>(dev_nodes, dev_buffer1, dev_buffer2, params);
  hipGetLastError();

  FLOAT3 * out = new FLOAT3[nbody];
  hipMemcpy(out, dev_buffer1, nbody*sizeof(FLOAT3), hipMemcpyDeviceToHost);

  for (int i = 0; i < nbody; i++) {
    std::cout << unc_pos[i].x << " " << out[i].x << " + " << unc_pos[i].y << " " << out[i].y << " + " << unc_pos[i].z << " " << out[i].z << '\n';
  }

  hipFree(dev_nodes);
  hipFree(dev_buffer1);
  hipFree(dev_buffer2);

}
